#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include<iostream>
#include<vector>
#include<algorithm>
#include<ctime>

#define THREAD_COUNT 320

using namespace std;

__global__ void random_outlist(bool *cugraph, int n, int *curand1, int *curand2, int f) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int degree, count = 3, temp;

	if (id < n) {
		degree = (curand1[id] + curand2[0])%f;
		if ((curand1[id] + curand2[1]) % 2)	degree *= (curand1[id] + curand2[2]) % 2 + 1;
		while (degree--) {
			temp = (curand1[id] + curand2[count++]) % n;
			if(temp != id)	cugraph[id*n + temp] = 1;
		}
	}

}

__global__ void get_lists(bool *cugraph, int *inlist, int *degree, int f, int n) {
	f = 2 * f + 1;
	int id = blockIdx.x * blockDim.x + threadIdx.x, start, count, start2;

	if (id < n) {
		start = id * f;
		start2 = id * n;
		count = 0;
		for (int i = 0; i < f; i++) {
			inlist[start + i] = -1;
		}
		degree[id] = 0;
		for (int i = 0; i < n; i++) {
			if (cugraph[start2 + i]) {
				degree[id]++;	
			}
		}

		for (int i = 0; i < n; i++) {
			if (cugraph[n*i + id]) {
				inlist[start + count++] = i;
			}
		}

	}

}

__global__ void random_surfer(bool *cugraph, int n, int visit_count) {
	int vis;

}

__global__ void single_rank(int *cuinlist, int *cudegree, double *pages, double *pages2, int f, int n, double d) {
	int id = blockDim.x * blockIdx.x + threadIdx.x, count = 0, start = id*f;
	double rank = (1 - d)/n, temp = 0;

	if (id < n) {
		while (cuinlist[start + count] != -1 && count < f) {
			temp += (pages[cuinlist[start + count]] / cudegree[cuinlist[start + count]]);
			count++;
		}
		temp *= d;
		rank += temp;
		pages2[id] = rank;
	}
}

__global__ void initial_ranks(double *pages, int n) {
	int id = blockDim.x*blockIdx.x + threadIdx.x;

	if (id < n)	pages[id] = (double)1 / n;

}

void pagerank(int *cuinlist, int *cudegree, double *cupages, double *cupages2, int n, int f) {
	double d = 0.85;
	int iter = 13;


	for (int i = 0; i < iter; i++) {
		if (i % 2) {
			single_rank<<<n/THREAD_COUNT + 1, THREAD_COUNT>>>(cuinlist, cudegree, cupages, cupages2, 2*f + 1, n, d);
		}
		else {
			single_rank << <n / THREAD_COUNT + 1, THREAD_COUNT >> >(cuinlist, cudegree, cupages2, cupages, 2*f + 1, n, d);
		}
	}
}

double* naive_pagerank(bool *graph, int n) {
	double *ranks = (double*)malloc(n * sizeof(double)); //Stores all the page ranks
	double d = 0.85; //Damping Factor = 0.85
	double temp;
	vector<int> outbounds = *new vector<int>(n, 0); //Number of outbound links for each vertex
													//List of inbound links for each vertex
	vector<vector<int>> inbounds = *new vector<vector<int>>(n, *new vector<int>());
	//Initializing the outbound, inbound and rank list.
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			if (graph[n*i + j]) {
				outbounds[i]++;
				inbounds[j].push_back(i);
			}
		}
		ranks[i] = 1 / n;
	}
	//Calculating pagerank. Number of iterations = 7
	for (int i = 0; i < 13; i++) {
		for (int j = 0; j < n; j++) {
			temp = 0;
			for (int k = 0; k < inbounds[j].size(); k++)
				temp += ranks[inbounds[j][k]] / outbounds[inbounds[j][k]];
			ranks[j] = (1 - d) / n + d * temp;
		}
	}
	return ranks;
}

float gpu_pagerank(bool *cugraph, int n, int f) {
	double *cupages, *cupages2, *pages;
	int *cuinlist, *cudegree;
	float gpu = 0;
	hipEvent_t start, stop;

	hipMallocManaged(&cuinlist, n*(2 * f + 1) * sizeof(int));
	hipMallocManaged(&cudegree, n * sizeof(int));

	hipMallocManaged(&cupages, n * sizeof(double));
	hipMallocManaged(&cupages2, n * sizeof(double));
	pages = (double*)malloc(n * sizeof(double));

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	get_lists << <n / THREAD_COUNT + 1, THREAD_COUNT >> > (cugraph, cuinlist, cudegree, f, n);
	initial_ranks << <n / THREAD_COUNT + 1, THREAD_COUNT >> > (cupages, n);
	pagerank(cuinlist, cudegree, cupages, cupages2, n, f);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(pages, cupages2, n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(cuinlist);
	hipFree(cudegree);
	hipFree(cupages);
	hipFree(cupages2);
	free(pages);

	return gpu;
}

double cpu_pagerank(bool *graph, int n) {
	clock_t begin, end;
	double *ranks = (double*)malloc(n * sizeof(double)), cpu;
	begin = clock();

	ranks = naive_pagerank(graph, n);

	end = clock();

	cpu = (double)(end - begin) / CLOCKS_PER_SEC;
	cpu *= 1000;
	free(ranks);
	return cpu;
}

int main() {
	int n = 4000, f = 80, count;
	bool *cugraph, *graph;
	int *rand1, *rand2, *curand1, *curand2, *degree;
	double *ranks, cpu;
	float gpu;
	rand1 = (int*)malloc(n * sizeof(int));
	rand2 = (int*)malloc(n * sizeof(int));

	hipMallocManaged(&curand1, n * sizeof(int));
	hipMallocManaged(&curand2, n * sizeof(int));

	for (int i = 0; i < n; i++) {
		rand1[i] = i;
		rand2[i] = i;
	}

	random_shuffle(rand1, rand1 + n - 1);
	random_shuffle(rand2, rand2 + n - 1);

	hipMemcpy(curand1, rand1, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(curand2, rand2, n * sizeof(int), hipMemcpyHostToDevice);
	//Need to check this logic

	hipMallocManaged(&cugraph, n * n * sizeof(bool));
	graph = (bool*)malloc(n * n * sizeof(bool));

	random_outlist<<<n/THREAD_COUNT + 1, THREAD_COUNT>>>(cugraph, n, curand1, curand2, f);

	hipFree(curand1);
	hipFree(curand2);
	free(rand1);
	free(rand2);

	hipMemcpy(graph, cugraph, n * n * sizeof(bool), hipMemcpyDeviceToHost);

	gpu = 0;
	for(int i = 0 ; i < 5 ; i++)	gpu += gpu_pagerank(cugraph, n, f);
	gpu /= 5;

	hipFree(cugraph);

	//for (int i = 0; i < n; i++)	cout << i << "\t" << pages[i] << "\n";

	cpu = 0;
	for(int i = 0 ; i < 5 ; i++)	cpu += cpu_pagerank(graph, n);
	cpu /= 5;

	cout << cpu << "ms\n";
	cout << gpu << "ms\n";
	cout << "Speed Up:    " << (cpu / ((double)gpu)) << "\n";

	//for (int i = 0; i < n; i++)	cout << i << "\t" << pages[i] << "\t" << ranks[i] << "\t" << "\n";

	return 0;
}